#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void doublevector(int* vec, int N)
{
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (idx < N) {
        vec[idx] *= 2;
    }
}

__global__ void init(int* vec, int N)
{
    int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (idx < N) {
        vec[idx] = idx;
    }
}

bool check(int* vec, int N)
{
    for (int i=0; i<N; ++i) {
        if (vec[i] != i*2) {
            return false;
        }
    }

    return true;
}

int main(int argc, char** argv)
{
    const int N = 4096;
    int* vec;

    // managed memory in cuda
    hipError_t result = hipMallocManaged(&vec, N * sizeof(int));
    if (result != hipSuccess) {
        printf("hipMalloc failed\n");
    }

    // initialize the vector on the gpu
    init<<<4, N/4>>>(vec, N);

    // launch a gpu kernel with 4 blocks and 1024 threads in each block.
    doublevector<<<4,N/4>>>(vec, N);

    // block the cpu for the gpu to finish execution
    hipDeviceSynchronize();

    if (!check(vec, N)) {
        printf("test failed\n");
    }

    hipFree(vec);
    return 0;
}
