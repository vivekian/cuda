#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloworld()
{
    printf("hello world\n");
}

int main(int argc, char** argv)
{
    // launch a gpu kernel
    helloworld<<<3,3>>>();

    // block the cpu for the gpu to finish execution
    hipDeviceSynchronize();
    return 0;
}
