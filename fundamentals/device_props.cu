#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

int main(int argc, char** argv)
{
   hipDeviceProp_t prop;
   hipGetDeviceProperties(&prop, 0);

   printf("Device name: %s\n", prop.name);
   printf("Memory clock rate (KHz) : %d\n", prop.memoryClockRate);
   printf("Memory bus width (bits) : %d\n", prop.memoryBusWidth);
   printf("Compute capability : %d.%d\n", prop.major, prop.minor);
   printf("Max threads per block : %d\n", prop.maxThreadsPerBlock);
   printf("Max threads per multiprocessor: %d\n", prop.maxThreadsPerMultiProcessor);
   printf("Multiprocessor count: %d\n", prop.multiProcessorCount);
   return 0;
}
