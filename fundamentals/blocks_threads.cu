#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void gpukernel()
{
    printf("%d %d %d %d\n", blockIdx.x, threadIdx.x, blockDim.x, gridDim.x);
}

int main(int argc, char** argv)
{
    // launch a gpu kernel with 3 blocks and 4 threads in each block.
    gpukernel<<<3,4>>>();

    // block the cpu for the gpu to finish execution
    hipDeviceSynchronize();
    return 0;
}
