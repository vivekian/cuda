#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void gpukernel()
{
    printf("%d %d\n", blockIdx.x, threadIdx.x);
}

int main(int argc, char** argv)
{
    // launch a gpu kernel with 3 blocks and 4 threads in each block.
    gpukernel<<<3,4>>>();

    // block the cpu for the gpu to finish execution
    hipDeviceSynchronize();
    return 0;
}
